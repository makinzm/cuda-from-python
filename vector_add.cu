#include <hip/hip_runtime.h>

extern "C" void vector_add(float *a, float *b, float *c, int n);

__global__ void vector_add_kernel(float *a, float *b, float *c, int n) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

void vector_add(float *a, float *b, float *c, int n) {
    float *d_a, *d_b, *d_c;
    size_t size = n * sizeof(float);

    // デバイスメモリの割り当て
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    // ホストからデバイスへデータを転送
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // カーネルの起動
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    vector_add_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, n);

    // デバイスからホストへ結果を転送
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // デバイスメモリの解放
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

